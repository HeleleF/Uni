#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"
#include <opencv2\opencv.hpp>

// Programmierkonzepte und Algorithmen
// Beleg Aufgabe 3
// Chris Rebbelin s0548921

/**
* CUDA kernel
*/
__global__ void switchGreenBlueKernel(const uchar3* const inImage, uchar3 *const outImage, const int w) {

	// 2D Array of Blocks with the same dimension as the inImg
	// copied from ProgKo_03-04_CUDA.pdf (page 57; 109-112)
	const long idx = blockIdx.y * w + blockIdx.x;

	// get current pixel
	const uchar3 pixel = inImage[idx];

	// save new pixel value
	// openCV uses BGR order (opposed to standard RGB)
	// so switching Blue and Green means switching the first two channels
	// TODO: Writing it like this causes problems on deepgreen (Why?)
	outImage[idx] = { pixel.y, pixel.x, pixel.z };
}

void switchWithCUDA(const char *inputFile, const char *outputFile) {

	printf("CUDA start\n");

	Timer timer;

	// 3 unsigned chars for 3 channels (alpha is not needed)
	uchar3 *host_inputImage, *host_outputImage, *dev_inputImage, *dev_outputImage;

	// read the input image with openCV
	// channel order is Blue Green Red !
	// (assuming we're only dealing with color images)
	cv::Mat inImg = cv::imread(inputFile, CV_LOAD_IMAGE_COLOR);

	// check for errors
	if (!inImg.data) {
		printf("Could not open or find the image");
		return;
	}

	// allocate memory for the output image
	// (CV_8UC3 is 3 channels with 8 byte each)
	cv::Mat outImg = cv::Mat(inImg.rows, inImg.cols, CV_8UC3);

	// get uchar3 pointer to both host images
	host_inputImage = inImg.ptr<uchar3>(0);
	host_outputImage = outImg.ptr<uchar3>(0);

	// calculate buffer size
	const size_t bufSize = sizeof(uchar3) * inImg.rows * inImg.cols;

	// set up device and print properties
	int dev = 0;
	CHECK(hipSetDevice(dev));

	hipDeviceProp_t prop;
	CHECK(hipGetDeviceProperties(&prop, dev));
	printf("Device %s\n", prop.name);
	printf("%d GPU processors\n", prop.multiProcessorCount);

	// allocate memory for device input image
	CHECK(hipMalloc((void**)&dev_inputImage, bufSize));

	// allocate memory for device output image
	CHECK(hipMalloc((void**)&dev_outputImage, bufSize));

	// copy host image to device image
	CHECK(hipMemcpy(dev_inputImage, host_inputImage, bufSize, hipMemcpyHostToDevice));

	// 1 Thread per Block, 2D Array of Blocks with the same dimension as the input image
	// copied from ProgKo_03-04_CUDA.pdf (page 57; 109-112)
	const int w = inImg.rows;
	const int h = inImg.cols;

	dim3 grid(w, h, 1);
	dim3 block(1, 1, 1);
	printf("Execution configure <<<(%d,%d), %d>>>\n", grid.x, grid.y, block.x);

	// start the timer
	timer.Start();

	// invoke the kernel
	switchGreenBlueKernel <<<grid, block>>> (dev_inputImage, dev_outputImage, w);

	// stop timer
	timer.Stop();
	printf("CUDA end. Time elapsed: %f ms.\n", timer.GetTime());

	CHECK(hipDeviceSynchronize());

	// copy the output image from device back to host
	CHECK(hipMemcpy(host_outputImage, dev_outputImage, bufSize, hipMemcpyDeviceToHost));

	// write the output inImg
	cv::imwrite(outputFile, outImg);

	cv::imshow("meddl1", inImg);
	cv::imshow("meddl2", outImg);

	cv::waitKey(0);

	// free device memory
	CHECK(hipFree(dev_inputImage));
	CHECK(hipFree(dev_outputImage));

	// reset device and exit
	CHECK(hipDeviceReset());
}

void switchSimple(const char *inputFile, const char *outputFile) {

	printf("CPU start\n");

	// read the input image with openCV
	cv::Mat inImg = cv::imread(inputFile, CV_LOAD_IMAGE_COLOR);

	if (!inImg.data) {
		printf("Could not open or find the image");
		return;
	}

	// allocate memory for the output image
	cv::Mat outImg = cv::Mat(inImg.rows, inImg.cols, CV_8UC3);

	// get uchar3 pointer to both host images
	uchar3 *inputPtr = inImg.ptr<uchar3>(0);
	uchar3 *outputPtr = outImg.ptr<uchar3>(0);

	//**** TIME THIS 

	for (int i = 0; i < inImg.rows * inImg.cols; i++) {

		// get current pixel	
		const uchar3 pixel = inputPtr[i];

		// save new value
		// TODO: Writing it like this causes problems on deepgreen (Why?)
		outputPtr[i] = { pixel.y, pixel.x, pixel.z };
	}

	//**** TIMER END 

	// write the output inImg
	cv::imwrite(outputFile, outImg);

	printf("CPU end. Elapsed Time: ms\n");

}

void switchWithOpenCV(const char *inputFile, const char *outputFile) {

	printf("OpenCV start\n");

	// read the input image with openCV
	cv::Mat inImg = cv::imread(inputFile, CV_LOAD_IMAGE_COLOR);

	if (!inImg.data) {
		printf("Could not open or find the image");
		return;
	}

	// allocate memory for the output image
	cv::Mat outImg = cv::Mat(inImg.rows, inImg.cols, CV_8UC3);	
	std::vector<cv::Mat> channels(3);

	//**** TIME THIS 

	// split bgr image into 3 channels
	cv::split(inImg, channels);

	// create new vector with switched channels
	std::vector<cv::Mat> switchedChannels(3);
	switchedChannels[0] = channels[1];
	switchedChannels[1] = channels[0];
	switchedChannels[2] = channels[2];

	// merge channels again
	cv::merge(switchedChannels, outImg);

	//**** TIMER END 

	// write the output inImg
	cv::imwrite(outputFile, outImg);

	printf("OpenCV end. Elapsed Time: ms\n");
}

/**
* main function
*
* Compile on deepgreen with:
* nvcc ./start.cu `pkg-config --cflags --libs opencv` -o main.out
*
* Start with: ./main.out testimages/dice.png testimages/output.png
*
* TODO: Benchmark all 3 functions (Time, Complexity, Effort) (How?)
* TODO: Find a better way for the kernel grid (?)
* TODO: Timer for non-CUDA stuff
*/
int main(int argc, char **argv) {

	// check for correct cmd args
	if (argc != 3) {
		printf("Usage: %s inputImg outputImg \n", argv[0]);
		return EXIT_FAILURE;
	}

	switchWithCUDA(argv[1], argv[2]);

	//switchWithOpenCV(argv[1], argv[2]);

	//switchSimple(argv[1], argv[2]);

	return EXIT_SUCCESS;
}